#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <cmath>

//#define TILE_WIDTH 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void init_array(float *A, int R, int C) {
    for (int i = 0; i < R; i++) {
        for (int j = 0; j < C; j++) {
            //A[i*C+j] = (float) rand() / RAND_MAX;
           A[i*C+j] = (i+j);
        }
    }
}

void matrix_multi_cpu(float *A, float *B, float *C, int N, int K, int M) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            float pval = 0;
            for (int k = 0; k < K; k++) {
                pval += A[i*K+k] * B[k*M + j];
            }
            C[i*M + j] = pval;
        }
    }
}

__global__ void tiled_matrix_multiplication_gpu(float *A, float *B, float *C, int N, int K, int M, 
                    unsigned int Ads_size, unsigned int Bds_size) {
    extern __shared__ float ABds[];
    float *Ads = (float *)ABds;
    float *Bds = (float *)ABds + Ads_size;

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int col = bx * blockDim.x + tx;
    int row = by * blockDim.y + ty;

    int TILE_WIDTH = blockDim.x;

    float p_val = 0;
    for (int ph = 0; ph < ceil(K/(float)TILE_WIDTH); ph++) {
        //Load part of data into shared memory
        if ((row < N) && (ph * TILE_WIDTH + tx) < K)
            Ads[ty * TILE_WIDTH + tx] = A[row * K + ph * TILE_WIDTH + tx];
        else Ads[ty * TILE_WIDTH + tx] =0.0f;
        if (((ty + ph * TILE_WIDTH) < K) && col < M)
            Bds[ty * TILE_WIDTH + tx] = B[(ty + ph * TILE_WIDTH) * M + col];
        else Bds[ty * TILE_WIDTH + tx] = 0.0f;
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            p_val += Ads[ty * TILE_WIDTH + k] * Bds[k * TILE_WIDTH + tx];
        }
        __syncthreads();
    }
    if (row < N && col < M)
        C[row*M+col] = p_val;
}


void print_matrix_results(float *A, float *B, float *C, int N, int K, int M) {
    printf("########################################\n");
    printf("Matrix A\n");
    for (int i = 0; i < N; i++) {
        for (int j =0; j < K; j++) {
            printf("%f ", A[i*K+j]);
        }
        printf("\n");
    }
    printf("########################################\n");
    printf("Matrix B\n");
    for (int i = 0; i < K; i++) {
        for (int j =0; j < M; j++) {
            printf("%f ", B[i*M+j]);
        }
        printf("\n");
    }
    printf("########################################\n");
    printf("Matrix C\n");
    for (int i = 0; i < N; i++) {
        for (int j =0; j < M; j++) {
            printf("%f ", C[i*M+j]);
        }
        printf("\n");
    }
}


bool validate_arrays(float *A, float *B, int N, int M) {
    bool is_matched = true;
    for (int i =0; i < N; i++) {
        for (int j =0; j < M; j++) {
            if (abs(A[i*M+j] - B[i*M+j]) > 1e-3) {
                printf("Array Mismatch at (%d, %d) - A: %f, B - %f\n", i, j, A[i*M+j], B[i*M+j]);
                is_matched = false;
            }
        }
    }
    return is_matched;
}


int main() {
    //time
    struct timeval t1, t2;
    // Define Host Arrays
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    // Define Array size
    int N = 3;
    int K = 2;
    int M = 3;

    // Allocate memory
    h_A = (float*)malloc(N * K * sizeof(float));
    h_B = (float*)malloc(K * M * sizeof(float));
    h_C_cpu = (float*)malloc(N * M * sizeof(float));
    h_C_gpu = (float*)malloc(N * M * sizeof(float));

    // Init array
    init_array(h_A, N , K);
    init_array(h_B, K , M);

    // #### GPU #####
    float *d_A, *d_B, *d_C;

    // Alloc memory
    hipMalloc((void**)&d_A, N * K * sizeof(float));
    hipMalloc((void**)&d_B, K * M * sizeof(float));
    hipMalloc((void**)&d_C, N * M * sizeof(float));

    // Copy to device
    hipMemcpy(d_A, h_A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * M * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    int width = (int)std::sqrt(devProp.maxThreadsPerBlock);

    printf("Width : %d\n", width);

    // Call kernel
    dim3 dim_block(width,width);
    dim3 dim_grid(ceil(M * 1.0f/width), ceil(N * 1.0f/width));
    size_t sharedMemSize = 2 * width * width * sizeof(float); 
    gettimeofday(&t1, 0);
    tiled_matrix_multiplication_gpu<<<dim_grid, dim_block, sharedMemSize>>>(d_A, d_B, d_C, N, K, M, width * width,width * width);
    //6. Check error from kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    gpuErrchk(hipDeviceSynchronize());
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("GPU Time to generate:  %3.1f ms \n", time);

    // Copy to host
    hipMemcpy(h_C_gpu, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);

    print_matrix_results(h_A, h_B, h_C_gpu, N, K, M);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}